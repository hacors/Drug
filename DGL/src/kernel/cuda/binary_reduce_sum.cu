/*!
 *  Copyright (c) 2019 by Contributors
 * \file kernel/cuda/binary_reduce_sum.cu
 * \brief CUDA kernels for binary reduce sum
 */
#include <dgl/runtime/device_api.h>

#include "../../runtime/cuda/cuda_common.h"
#include "./binary_reduce_impl.cuh"
#include "./backward_binary_reduce_impl.cuh"
#include "../utils.h"
#include "../csr_interface.h"

using minigun::advance::RuntimeConfig;

namespace dgl {
namespace kernel {
namespace cuda {
// specialization for cusparse

template <typename DType>
hipsparseStatus_t Xcsrmm2(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const DType* alpha, const hipsparseMatDescr_t descrA,
    const DType* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const DType* B, int ldb, const DType* beta, DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPSPARSE_STATUS_EXECUTION_FAILED;
}

template <>
hipsparseStatus_t Xcsrmm2<float>(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const float* alpha, const hipsparseMatDescr_t descrA,
    const float* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const float* B, int ldb, const float* beta, float* C, int ldc) {
  return hipsparseScsrmm2(handle, transA, transB, m, n, k, nnz,
      alpha, descrA, csrValA, csrRowPtrA, csrColIndA,
      B, ldb, beta, C, ldc);
}

template <>
hipsparseStatus_t Xcsrmm2<double>(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const double* alpha, const hipsparseMatDescr_t descrA,
    const double* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const double* B, int ldb, const double* beta, double* C, int ldc) {
  return hipsparseDcsrmm2(handle, transA, transB, m, n, k, nnz,
      alpha, descrA, csrValA, csrRowPtrA, csrColIndA,
      B, ldb, beta, C, ldc);
}

template <typename DType>
hipblasStatus_t Xgeam(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const DType* alpha, const DType* A, int lda,
    const DType* beta, const DType* B, int ldb,
    DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPBLAS_STATUS_EXECUTION_FAILED;
}

template <>
hipblasStatus_t Xgeam<float>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const float* alpha, const float* A, int lda,
    const float* beta, const float* B, int ldb,
    float* C, int ldc) {
  return hipblasSgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

template <>
hipblasStatus_t Xgeam<double>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const double* alpha, const double* A, int lda,
    const double* beta, const double* B, int ldb,
    double* C, int ldc) {
  return hipblasDgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

template <typename DType>
void CusparseCsrmm2(
    const RuntimeConfig& rtcfg,
    const aten::CSRMatrix& csr,
    const DType* B_data, DType* C_data,
    int x_length) {
  // We use csrmm2 to perform following operation:
  // C = A x B, where A is a sparse matrix in csr format, B is the dense matrix for node
  // feature tensor. However, since cusparse only supports column-major, while our tensor
  // is stored in row-major, the actual computation is:
  // C = trans(A x trans(B)).
  // Currently, we use cublasXgeam to implement transposition and allocate intermediate
  // workspace memory for this.
  const int m = csr.num_rows;
  const int n = x_length;
  const int k = csr.num_cols;
  const int nnz = csr.indices->shape[0];
  const DType alpha = 1.0;
  const DType beta = 0.0;
  // device
  auto device = runtime::DeviceAPI::Get(rtcfg.ctx);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, rtcfg.stream));
  // allocate matrix for temporary transposed output
  DType* trans_out = static_cast<DType*>(device->AllocWorkspace(rtcfg.ctx, m * n * sizeof(DType)));
  // all one data array
  DType* valptr = static_cast<DType*>(device->AllocWorkspace(rtcfg.ctx, nnz * sizeof(DType)));
  utils::Fill<kDLGPU>(rtcfg.ctx, valptr, nnz, static_cast<DType>(1.));
  hipsparseMatDescr_t descr;
  CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
  CUSPARSE_CALL(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CUSPARSE_CALL(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));
  CUSPARSE_CALL(Xcsrmm2<DType>(
      thr_entry->cusparse_handle,
      HIPSPARSE_OPERATION_NON_TRANSPOSE,
      HIPSPARSE_OPERATION_TRANSPOSE,
      m, n, k, nnz, &alpha,
      descr, valptr,
      static_cast<int32_t*>(csr.indptr->data),
      static_cast<int32_t*>(csr.indices->data),
      B_data, n, &beta, trans_out, m));
  device->FreeWorkspace(rtcfg.ctx, valptr);
  // transpose the output matrix
  if (!thr_entry->cublas_handle) {
    CUBLAS_CALL(hipblasCreate(&(thr_entry->cublas_handle)));
  }
  CUBLAS_CALL(hipblasSetStream(thr_entry->cublas_handle, rtcfg.stream));
  CUBLAS_CALL(Xgeam<DType>(
      thr_entry->cublas_handle,
      HIPBLAS_OP_T,
      HIPBLAS_OP_N,
      n, m,
      &alpha, trans_out, m,
      &beta, nullptr, n,
      C_data, n));
  device->FreeWorkspace(rtcfg.ctx, trans_out);
}

// forward

template <typename DType>
void FallbackCallBinaryReduce(
    const RuntimeConfig& rtcfg,
    const CSRWrapper& graph,
    GData<int32_t, DType>* gdata) {
  constexpr int XPU = kDLGPU;
  typedef int32_t Idx;
  typedef SelectSrc LeftSelector;
  typedef SelectNone RightSelector;
  typedef BinaryUseLhs<DType> BinaryOp;
  typedef ReduceSum<kDLGPU, DType> Reducer;
  typedef cuda::FunctorsTempl<Idx, DType, LeftSelector,
                        RightSelector, BinaryOp, Reducer>
          Functors;
  typedef cuda::BinaryReduce<Idx, DType, Functors> UDF;
  // csr
  auto outcsr = graph.GetOutCSRMatrix();
  minigun::Csr<Idx> csr = utils::CreateCsr<Idx>(outcsr.indptr, outcsr.indices);
  // If the user-given mapping is none and the target is edge data, we need to
  // replace the mapping by the edge ids in the csr graph so that the edge
  // data is correctly read/written.
  if (LeftSelector::target == binary_op::kEdge && gdata->lhs_mapping == nullptr) {
    gdata->lhs_mapping = static_cast<Idx*>(outcsr.data->data);
  }
  if (RightSelector::target == binary_op::kEdge && gdata->rhs_mapping == nullptr) {
    gdata->rhs_mapping = static_cast<Idx*>(outcsr.data->data);
  }
  if (OutSelector<Reducer>::Type::target == binary_op::kEdge
      && gdata->out_mapping == nullptr) {
    gdata->out_mapping = static_cast<Idx*>(outcsr.data->data);
  }
  // TODO(minjie): allocator
  minigun::advance::Advance<XPU, Idx, cuda::AdvanceConfig, GData<Idx, DType>, UDF>(
        rtcfg, csr, gdata, minigun::IntArray1D<Idx>());
}

template <typename DType>
void FallbackCallBackwardBinaryReduce(
    const RuntimeConfig& rtcfg,
    const CSRWrapper& graph,
    BackwardGData<int32_t, DType>* gdata) {
  constexpr int XPU = kDLGPU;
  constexpr int Mode = binary_op::kGradLhs;
  typedef int32_t Idx;
  typedef SelectSrc LeftSelector;
  typedef SelectNone RightSelector;
  typedef BinaryUseLhs<DType> BinaryOp;
  typedef ReduceSum<kDLGPU, DType> Reducer;
  // For backward computation, we use reverse csr and switch dst and src.
  // This benefits the most common src_op_edge or copy_src case, because the
  // gradients of src are now aggregated into destination buffer to reduce
  // competition of atomic add.
  auto incsr = graph.GetInCSRMatrix();
  minigun::Csr<Idx> csr = utils::CreateCsr<Idx>(incsr.indptr, incsr.indices);
  typedef cuda::BackwardFunctorsTempl<Idx, DType,
          typename SwitchSrcDst<LeftSelector>::Type,
          typename SwitchSrcDst<RightSelector>::Type,
          BinaryOp, Reducer> Functors;
  typedef cuda::BackwardBinaryReduce<Mode, Idx, DType, Functors> UDF;
  // If the user-given mapping is none and the target is edge data, we need to
  // replace the mapping by the edge ids in the csr graph so that the edge
  // data is correctly read/written.
  if (LeftSelector::target == binary_op::kEdge
      && gdata->lhs_mapping == nullptr) {
    gdata->lhs_mapping = static_cast<Idx*>(incsr.data->data);
  }
  if (RightSelector::target == binary_op::kEdge
      && gdata->rhs_mapping == nullptr) {
    gdata->rhs_mapping = static_cast<Idx*>(incsr.data->data);
  }
  if (OutSelector<Reducer>::Type::target == binary_op::kEdge
      && gdata->out_mapping == nullptr) {
    gdata->out_mapping = static_cast<Idx*>(incsr.data->data);
  }
  // TODO(minjie): allocator
  minigun::advance::Advance<XPU, Idx, cuda::AdvanceConfig, BackwardGData<Idx, DType>, UDF>(
        rtcfg, csr, gdata, minigun::IntArray1D<Idx>());
}

}  // namespace cuda

template <>
void CallBinaryReduce<kDLGPU, int32_t, float, SelectSrc, SelectNone,
                      BinaryUseLhs<float>, ReduceSum<kDLGPU, float>>(
    const RuntimeConfig& rtcfg,
    const CSRWrapper& graph,
    GData<int32_t, float>* gdata) {
  if (gdata->lhs_mapping || gdata->rhs_mapping || gdata->out_mapping) {
    cuda::FallbackCallBinaryReduce<float>(rtcfg, graph, gdata);
  } else {
    // cusparse use rev csr for csrmm
    auto csr = graph.GetInCSRMatrix();
    cuda::CusparseCsrmm2(rtcfg, csr, gdata->lhs_data, gdata->out_data,
        gdata->x_length);
  }
}

template <>
void CallBinaryReduce<kDLGPU, int32_t, double, SelectSrc, SelectNone,
                      BinaryUseLhs<double>, ReduceSum<kDLGPU, double>>(
    const RuntimeConfig& rtcfg,
    const CSRWrapper& graph,
    GData<int32_t, double>* gdata) {
  if (gdata->lhs_mapping || gdata->rhs_mapping || gdata->out_mapping) {
    cuda::FallbackCallBinaryReduce<double>(rtcfg, graph, gdata);
  } else {
    // cusparse use rev csr for csrmm
    auto csr = graph.GetInCSRMatrix();
    cuda::CusparseCsrmm2(rtcfg, csr, gdata->lhs_data, gdata->out_data,
        gdata->x_length);
  }
}

// backward

template <>
void CallBackwardBinaryReduce<kDLGPU, binary_op::kGradLhs, int32_t, float,
                              SelectSrc, SelectNone,
                              BinaryUseLhs<float>, ReduceSum<kDLGPU, float>>(
    const RuntimeConfig& rtcfg,
    const CSRWrapper& graph,
    BackwardGData<int32_t, float>* gdata) {
  if (gdata->lhs_mapping || gdata->rhs_mapping || gdata->out_mapping) {
    cuda::FallbackCallBackwardBinaryReduce<float>(rtcfg, graph, gdata);
  } else {
    auto csr = graph.GetOutCSRMatrix();
    cuda::CusparseCsrmm2(rtcfg, csr, gdata->grad_out_data, gdata->grad_lhs_data,
        gdata->x_length);
  }
}

template <>
void CallBackwardBinaryReduce<kDLGPU, binary_op::kGradLhs, int32_t, double,
                              SelectSrc, SelectNone,
                              BinaryUseLhs<double>, ReduceSum<kDLGPU, double>>(
    const RuntimeConfig& rtcfg,
    const CSRWrapper& graph,
    BackwardGData<int32_t, double>* gdata) {
  if (gdata->lhs_mapping || gdata->rhs_mapping || gdata->out_mapping) {
    cuda::FallbackCallBackwardBinaryReduce<double>(rtcfg, graph, gdata);
  } else {
    auto csr = graph.GetOutCSRMatrix();
    cuda::CusparseCsrmm2(rtcfg, csr, gdata->grad_out_data, gdata->grad_lhs_data,
        gdata->x_length);
  }
}

// generate definitions

#define REDUCER ReduceSum
#define XPU kDLGPU
#define IDX int32_t

EVAL(GEN_DTYPE, GEN_OP_TARGET, GEN_DEFINE);
EVAL(GEN_BACKWARD_MODE, GEN_DTYPE, GEN_OP_TARGET, GEN_BACKWARD_DEFINE);

}  // namespace kernel
}  // namespace dgl
