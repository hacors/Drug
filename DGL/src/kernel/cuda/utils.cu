#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2019 by Contributors
 * \file kernel/cuda/utils.cu
 * \brief Utility function implementations on CUDA
 */
#include "../../runtime/cuda/cuda_common.h"
#include "../utils.h"

namespace dgl {
namespace kernel {
namespace utils {

template <typename DType>
__global__ void _FillKernel(DType* ptr, size_t length, DType val) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    ptr[tx] = val;
    tx += stride_x;
  }
}

template <int XPU, typename DType>
void Fill(const DLContext& ctx, DType* ptr, size_t length, DType val) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  int nt = utils::FindNumThreads(length, 1024);
  int nb = (length + nt - 1) / nt;
  _FillKernel<<<nb, nt, 0, thr_entry->stream>>>(ptr, length, val);
}

template void Fill<kDLGPU, float>(const DLContext& ctx, float* ptr, size_t length, float val);
template void Fill<kDLGPU, double>(const DLContext& ctx, double* ptr, size_t length, double val);

}  // namespace utils
}  // namespace kernel
}  // namespace dgl
